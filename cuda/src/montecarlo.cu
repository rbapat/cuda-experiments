#include "hip/hip_runtime.h"
#include "montecarlo.cuh"

namespace montecarlo {

    __global__ void estimationKernel(hiprandState* states, int* blocksInCircle) {
        __shared__ bool in_circle[1024]; 

        ulong idx = threadIdx.x + blockIdx.x * blockDim.x;

        float x = 2 * hiprand_uniform(states + idx) - 1;
        float y = 2 * hiprand_uniform(states + idx) - 1;
        in_circle[threadIdx.x] = x*x + y*y <= 1;

        __syncthreads();

        if (threadIdx.x == 0) {
            int sum = 0;
            for (int i = 0; i < blockDim.x; i++) {
                sum += in_circle[i] ? 1 : 0;
            }

            blocksInCircle[blockIdx.x] = sum;
        }
    }

    __global__ void populateRandStates(hiprandState* states) {
        ulong idx = threadIdx.x + blockIdx.x * blockDim.x;
        hiprand_init(600382584, idx, 0, states + idx);
    }

    float estimate_pi(int numSamples, int tpb) {
        if (numSamples % tpb != 0) {
            throw std::runtime_error("numSamples must be divisible by tpb");
        }

        int numBlocks = numSamples / tpb;
        int numBytes = numBlocks * sizeof(int);
        hiprandState* randStates;
        hipMalloc(&randStates, sizeof(hiprandState) * 1024);
        populateRandStates<<<32, 32>>>(randStates);
        hipDeviceSynchronize();

        int* deviceBlocksInCircle = nullptr;
        hipError_t mallocRes = hipMalloc(&deviceBlocksInCircle, numBytes);
        hipError_t fillRes = hipMemset(deviceBlocksInCircle, 0, numBytes);
        if (mallocRes != hipSuccess || fillRes != hipSuccess) {
            throw std::runtime_error("Failed to allocate deviceBlocksInCircle buffer");
        }

        estimationKernel<<<numBlocks, tpb>>>(randStates, deviceBlocksInCircle);
        hipDeviceSynchronize();
        
        int* hostBlocksInCircle = (int*)malloc(numBytes);
        if (hostBlocksInCircle == 0) {
            throw std::runtime_error("Failed to malloc that many bytes");
        }
        if (hipMemcpy(hostBlocksInCircle, deviceBlocksInCircle, numBytes, hipMemcpyDeviceToHost) != hipSuccess) {
            printf("%s\n", hipGetErrorString(hipGetLastError()));
            throw std::runtime_error("Failed to copy device to host blocks in circle");
        }

        int sum = 0.f;
        for (int i = 0; i < numBlocks; i++) {
            sum += hostBlocksInCircle[i];
        }

        return 4.f * (float)sum / numSamples;
    }


    float time_pi_estimate(int numReps, int numSamples, int tpb) {
        float avgTimeUs = 0;
        
        for (int i = 0; i < numReps; i++) {
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            
            estimate_pi(numSamples, tpb);

            hipEventRecord(start);

            estimate_pi(numSamples, tpb);

            hipEventRecord(stop);
            hipEventSynchronize(stop);

            
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            avgTimeUs += milliseconds * 1000 / numReps;
        }

        return avgTimeUs;
    }
}