#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

#include "montecarlo.cuh"
namespace montecarlo {

__global__ void populateRandStates(hiprandState* randStates) {
  ulong idx = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(600382584, idx, 0, randStates + idx);
}

__global__ void estimate_pi(hiprandState* randStates, int* out_arr,
                            int numReps) {
  ulong idx = threadIdx.x + blockIdx.x * blockDim.x;

  int inCircle = 0;
  for (size_t rep = 0; rep < numReps; rep++) {
    float x = 2 * hiprand_uniform(randStates + idx) - 1;
    float y = 2 * hiprand_uniform(randStates + idx) - 1;

    if (x * x + y * y <= 1) inCircle++;
  }

  out_arr[idx] = inCircle;
}

Naive::Naive(int _numSamples, int _blocksPerGrid, int _threadsPerBlock)
    : numSamples(_numSamples),
      blocksPerGrid(_blocksPerGrid),
      threadsPerBlock(_threadsPerBlock) {
  cudaCheckError(hipMalloc(
      &randStates, sizeof(hiprandState) * blocksPerGrid * threadsPerBlock));

  const dim3 gridSize(blocksPerGrid);
  const dim3 blockSize(threadsPerBlock);

  populateRandStates<<<gridSize, blockSize>>>(randStates);
}

std::string_view Naive::getName() { return "Monte Carlo Pi Estimation"; }

float Naive::naiveReduce(int* simSums) {
  int* simSumsHost =
      (int*)malloc(blocksPerGrid * threadsPerBlock * sizeof(int));
  cudaCheckError(hipMemcpy(simSumsHost, simSums,
                            blocksPerGrid * threadsPerBlock * sizeof(int),
                            hipMemcpyDeviceToHost));

  int sum = 0;
  for (size_t i = 0; i < blocksPerGrid * threadsPerBlock; i++) {
    sum += simSumsHost[i];
  }

  free(simSumsHost);
  return 4.f * sum / numSamples;
}

float Naive::cubReduce(int* simSums) {
  int* deviceOut;
  cudaCheckError(hipMalloc(&deviceOut, sizeof(int)));

  void* tempStorage = NULL;
  size_t tempStorageSize = 0;
  const int numItems = blocksPerGrid * threadsPerBlock;

  CubDebugExit(hipcub::DeviceReduce::Sum(tempStorage, tempStorageSize, simSums,
                                      deviceOut, numItems));

  cudaCheckError(hipMalloc(&tempStorage, tempStorageSize));

  CubDebugExit(hipcub::DeviceReduce::Sum(tempStorage, tempStorageSize, simSums,
                                      deviceOut, numItems));

  int hostOut = 0;
  cudaCheckError(
      hipMemcpy(&hostOut, deviceOut, sizeof(int), hipMemcpyDeviceToHost));

  hipFree(deviceOut);
  hipFree(tempStorage);

  return 4.f * hostOut / (numSamples * numItems);
}

void Naive::calculate() {
  int* simSums = nullptr;
  cudaCheckError(
      hipMalloc(&simSums, blocksPerGrid * threadsPerBlock * sizeof(int)));

  const int repsPerThread = numSamples;
  const dim3 gridSize(blocksPerGrid, 1, 1);
  const dim3 blockSize(threadsPerBlock, 1, 1);

  estimate_pi<<<gridSize, blockSize>>>(randStates, simSums, repsPerThread);

  float pi = cubReduce(simSums);
  hipFree(simSums);
}

Naive::~Naive() { hipFree(randStates); }
}  // namespace montecarlo