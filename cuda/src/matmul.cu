#include "hip/hip_runtime.h"
#include "matmul.cuh"

namespace matmul {

    namespace naive {
        __global__ void matmul_kernel(float* mat1, float* mat2, float *out, int matrixSize) {
            int idxX = blockIdx.x * blockDim.x + threadIdx.x;
            int idxY = blockIdx.y * blockDim.y + threadIdx.y;

            float total = 0;
            for (int i = 0; i < matrixSize; i++) {
                total += mat1[matrixSize * idxY + i] * mat2[matrixSize * idxX + i];
            }

            out[idxY * matrixSize + idxX] = total;
        }

        float time_execution(int numTimes, int matrixSize, int tpb) {
            if (matrixSize % tpb != 0) {
                throw std::runtime_error("matrixSize must be a multiple of threadsPerBlock");
            }

            float* mat1 = matrix::generateRandomMatrixCuda(matrixSize);
            float* mat2 = matrix::generateRandomMatrixCuda(matrixSize);
            float* out = matrix::allocateEmptyMatrixCuda(matrixSize);

            float avgTimeUs = 0;

            for (int i = 0; i < numTimes; i++) {
                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);
                
                dim3 gridDim(matrixSize / tpb, matrixSize / tpb);
                dim3 blockDim(tpb, tpb);

                hipEventRecord(start);

                matmul_kernel<<<gridDim, blockDim>>>(mat1, mat2, out, matrixSize);

                hipEventRecord(stop);
                hipEventSynchronize(stop);

                
                float milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start, stop);
                avgTimeUs += milliseconds * 1000 / numTimes;
            }

            hipFree(mat1);
            hipFree(mat2);
            hipFree(out);

            return avgTimeUs;
        }
    }

    namespace cublas {
        float time_execution(int numTimes, int matrixSize, int tpb) {
            if (matrixSize % tpb != 0) {
                throw std::runtime_error("matrixSize must be a multiple of threadsPerBlock");
            }

            float* mat1 = matrix::generateRandomMatrixCuda(matrixSize);
            float* mat2 = matrix::generateRandomMatrixCuda(matrixSize);
            float* out = matrix::allocateEmptyMatrixCuda(matrixSize);

            const float alpha = 1.0f;
            const float beta  = 0.0f;

            float avgTimeUs = 0;
            

            hipblasHandle_t handle;
            if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS) {
                throw std::runtime_error("Failed to create cublas handle");
            }

            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrixSize, matrixSize, matrixSize, &alpha, mat2, matrixSize, mat1, matrixSize, &beta, out, matrixSize);

            for (int i = 0; i < numTimes; i++) {
                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);
                
                dim3 gridDim(matrixSize / tpb, matrixSize / tpb);
                dim3 blockDim(tpb, tpb);

                hipEventRecord(start);

                hipblasStatus_t res = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrixSize, matrixSize, matrixSize, &alpha, mat2, matrixSize, mat1, matrixSize, &beta, out, matrixSize);

                hipEventRecord(stop);
                hipEventSynchronize(stop);

                if (res != HIPBLAS_STATUS_SUCCESS) {
                    std::cout << "Failed to do hipblasSgemm at iteration " << i << " with error " << hipGetErrorString(hipGetLastError()) << std::endl;
                }
                
                float milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start, stop);
                avgTimeUs += milliseconds * 1000 / numTimes;
            }

            hipFree(mat1);
            hipFree(mat2);
            hipFree(out);

            return avgTimeUs;
        }
    }
}