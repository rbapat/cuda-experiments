#include "hip/hip_runtime.h"
#include "matmul.cuh"

namespace matmul {

    namespace naive {
        __global__ void matmul_kernel(float* mat1, float* mat2, float *out, int matrixSize) {
            int idxX = blockIdx.x * blockDim.x + threadIdx.x;
            int idxY = blockIdx.y * blockDim.y + threadIdx.y;

            float total = 0;
            for (int i = 0; i < matrixSize; i++) {
                total += mat1[matrixSize * idxY + i] * mat2[matrixSize * idxX + i];
            }

            out[idxY * matrixSize + idxX] = total;
        }

        float time_execution(int numTimes, int matrixSize, int tpb) {
            if (matrixSize % tpb != 0) {
                throw std::runtime_error("matrixSize must be a multiple of threadsPerBlock");
            }

            float* mat1 = matrix::generateRandomMatrixCuda(matrixSize);
            float* mat2 = matrix::generateRandomMatrixCuda(matrixSize);
            float* out = matrix::allocateEmptyMatrixCuda(matrixSize);

            float avgTimeUs = 0;

            for (int i = 0; i < numTimes; i++) {
                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);
                
                dim3 gridDim(matrixSize / tpb, matrixSize / tpb);
                dim3 blockDim(tpb, tpb);

                hipEventRecord(start);

                matmul_kernel<<<gridDim, blockDim>>>(mat1, mat2, out, matrixSize);

                hipEventRecord(stop);
                hipEventSynchronize(stop);

                
                float milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start, stop);
                avgTimeUs += milliseconds * 1000 / numTimes;
            }

            hipFree(mat1);
            hipFree(mat2);
            hipFree(out);

            return avgTimeUs;
        }
    }
}